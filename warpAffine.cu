
#include<hip/hip_runtime.h>

#define min(a, b)  ((a) < (b) ? (a) : (b))
#define num_threads   512
typedef unsigned char uint8_t;

struct Size{
    int width = 0, height = 0;

    Size() = default;
    Size(int w, int h)
    :width(w), height(h){}
};

struct AffineMatrix{
    float i2d[6]; float d2i[6];
    void invertAffineTransform(float imat[6], float omat[6]){
        float i00 = imat[0];  float i01 = imat[1];  float i02 = imat[2];
        float i10 = imat[3];  float i11 = imat[4];  float i12 = imat[5];

        // 计算行列式
        float D = i00 * i11 - i01 * i10;
        D = D != 0 ? 1.0 / D : 0;

        // 计算剩余的伴随矩阵除以行列式
        float A11 = i11 * D;
        float A22 = i00 * D;
        float A12 = -i01 * D;
        float A21 = -i10 * D;
        float b1 = -A11 * i02 - A12 * i12;
        float b2 = -A21 * i02 - A22 * i12;
        omat[0] = A11;  omat[1] = A12;  omat[2] = b1;
        omat[3] = A21;  omat[4] = A22;  omat[5] = b2;
    }
    void compute(const Size& from , const Size& to){
    float scale_x = to.width / (float)from.width;

    float scale_y = to.height / (float)from.height;
    float scale = min(scale_x, scale_y);

    i2d[0] = scale;  i2d[1] = 0;  i2d[2] = (-scale * from.width  + to.width + scale  - 1) * 0.5;
    i2d[3] = 0;  i2d[4] = scale;  i2d[5] = (-scale * from.height + to.height + scale - 1) * 0.5;
    invertAffineTransform(i2d, d2i);

    }

};
   
__device__ void affine_project(float* matrix, int x, int y, float* proj_x, float* proj_y){
    *proj_x = matrix[0] *x + matrix[1] *y + matrix[2] +0.5f;
    *proj_y = matrix[3] *x + matrix[4] *y + matrix[5] +0.5f;
}
   
__global__ void warp_affine_bilinear_kernel(uint8_t* src, int src_line_size,int src_width, int src_height,
                                            uint8_t* dst,int dst_line_size, int dst_width, int dst_height,
                                            uint8_t fill_value, AffineMatrix matrix){ 

    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if(dx>=dst_width || dy >= dst_height) return;
    float src_x =0;float src_y = 0;

 

    affine_project(matrix.d2i,dx,dy,&src_x,&src_y);


    // float c0 = 114;float c1=114;float c2=114;
    uint8_t  c0 = 114 ,c1=114,c2=114;


    if(src_x <-1 || src_x >= src_width || src_y<=-1 || src_y>= src_height){
    }else{
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);  
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_values[] = {fill_value,fill_value,fill_value};
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy*hx;
        float w2= hy*lx;
        float w3= ly*hx;
        float w4= ly*lx;

        uint8_t* v1 = const_values;
        uint8_t* v2 = const_values;
        uint8_t* v3 = const_values;
        uint8_t* v4 = const_values;

        if(y_low>=0){
            if(x_low >= 0)
            v1 = src + y_low*src_line_size + x_low*3;
            if(x_high <src_width)
            v2 = src + y_low*src_line_size + x_high*3;
            
        }
        if(y_high < src_height){
            if(x_low >= 0)
            v3 = src + y_high*src_line_size + x_low*3;
            if(x_high <src_width)
            v4 = src + y_high*src_line_size + x_high*3;  
                  }
        // c0 = min(floorf(w1*v1[0] + w2*v2[0] + w3*v3[0]+w4*v4[0] +0.5f),255);
        // c1 = min(floorf(w1*v1[1] + w2*v2[1] + w3*v3[1]+w4*v4[1] +0.5f),255);
        // c2 = min(floorf(w1*v1[2] + w2*v2[2] + w3*v3[2]+w4*v4[2] +0.5f),255);
        c0 = floorf(w1*v1[0] + w2*v2[0] + w3*v3[0]+w4*v4[0] +0.5f);
        c1 = floorf(w1*v1[1] + w2*v2[1] + w3*v3[1]+w4*v4[1] +0.5f);
        c2 = floorf(w1*v1[2] + w2*v2[2] + w3*v3[2]+w4*v4[2] +0.5f);

    }

    // #把计算好的值，放到目标图像对应的位置
    uint8_t* pdst = dst + dy*dst_line_size + dx*3;
    pdst[0]=  c0;
    pdst[1]=  c1;
    pdst[2]=  c2;
}

void warp_affine_bilinear(
       uint8_t* src, int src_line_size, int src_width, int src_height, 
    uint8_t* dst, int dst_line_size, int dst_width, int dst_height, 
	uint8_t fill_value){
    dim3 block_size(32,32);
    dim3 grid_size((dst_width + 31)/32,(dst_height +31)/32);
    AffineMatrix affine;
    affine.compute(Size(src_width,src_height),Size(dst_width,dst_height));

    warp_affine_bilinear_kernel<<< grid_size,block_size,0,nullptr>>>(        
        src, src_line_size, src_width, src_height,
        dst, dst_line_size, dst_width, dst_height,
        fill_value,affine);

}



__global__ void warp_affine_bilinear_kernel1(uint8_t* src, int src_line_size,int src_width, int src_height,
                                            uint8_t* dst,int dst_line_size, int dst_width, int dst_height,
                                            uint8_t fill_value, AffineMatrix matrix){ 

    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if(dx>=dst_width || dy >= dst_height) return;
    float src_x =0;float src_y = 0;

 

    affine_project(matrix.d2i,dx,dy,&src_x,&src_y);


    // float c0 = 114;float c1=114;float c2=114;
    uint8_t  c0 = 114 ;


    if(src_x <-1 || src_x >= src_width || src_y<=-1 || src_y>= src_height){
    }else{
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);  
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_values= fill_value;
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy*hx;
        float w2= hy*lx;
        float w3= ly*hx;
        float w4= ly*lx;

        uint8_t* v1 = &const_values;
        uint8_t* v2 = &const_values;
        uint8_t* v3 = &const_values;
        uint8_t* v4 = &const_values;

        if(y_low>=0){
            if(x_low >= 0)
            v1 = src + y_low*src_line_size + x_low;
            if(x_high <src_width)
            v2 = src + y_low*src_line_size + x_high;
            
        }
        if(y_high < src_height){
            if(x_low >= 0)
            v3 = src + y_high*src_line_size + x_low;
            if(x_high <src_width)
            v4 = src + y_high*src_line_size + x_high;  
                  }

        c0 = floorf(w1*v1[0] + w2*v2[0] + w3*v3[0]+w4*v4[0] +0.5f);


    }

    // #把计算好的值，放到目标图像对应的位置
    uint8_t* pdst = dst + dy*dst_line_size + dx;
    pdst[0]=  c0;
    }

void warp_affine_bilinear1(
       uint8_t* src, int src_line_size, int src_width, int src_height, 
    uint8_t* dst, int dst_line_size, int dst_width, int dst_height, 
	uint8_t fill_value){
    dim3 block_size(32,32);
    dim3 grid_size((dst_width + 31)/32,(dst_height +31)/32);
    AffineMatrix affine;
    affine.compute(Size(src_width,src_height),Size(dst_width,dst_height));

    warp_affine_bilinear_kernel1<<< grid_size,block_size,0,nullptr>>>(        
        src, src_line_size, src_width, src_height,
        dst, dst_line_size, dst_width, dst_height,
        fill_value,affine);

}
